#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <chrono> 
#include <stdio.h>
#include <GLFW/glfw3.h>
#include <math.h>
#include <iostream>
#include <stdlib.h> 
#include <vector>
#include <windows.h>
using namespace std;
using namespace std::chrono;
#define SCREEN_WIDTH 1600
#define SCREEN_HEIGHT 1000
#define NUMBER_BALL 128

__device__ float G = 6.67430e-11;
__device__ float M = 1e11;
__device__ float elastic = 0.85;
__device__ float R = 10;
__device__ float t = 1;
static float ballsize = 20;
void init(GLfloat arg[])
{
	for (int i = 0; i < NUMBER_BALL * 2; i+=2)
	{arg[i] = rand() % 1300 + 150;
		arg[i+1] = rand() % 700 + 150;
	}
}
void initV(GLfloat arg[])
{

	for (int n = 0; n < NUMBER_BALL * 2; n++)
	{
		arg[n] = rand() % 6-3;
	}
}
__global__
void collisionCudaShared(GLfloat *X, GLfloat*V)
{
	__shared__ GLfloat s[NUMBER_BALL * 4];
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	if (i < NUMBER_BALL * 2)
	{
		s[i] = X[i];
		s[i + 1] = X[i + 1];
		s[i + NUMBER_BALL * 2] = V[i];
		s[i + 1 + NUMBER_BALL * 2] = V[i + 1];

		if (s[i] <= R)
		{
			s[NUMBER_BALL * 2 + i] = -s[NUMBER_BALL * 2 + i] * elastic;
			s[i] = R;
		}
		if (s[i] >= SCREEN_WIDTH - R)
		{
			s[NUMBER_BALL * 2 + i] = -s[NUMBER_BALL * 2 + i] * elastic;
			s[i] = SCREEN_WIDTH - R;
		}
		if (s[i + 1] <= R)
		{
			s[NUMBER_BALL * 2 + i + 1] = -s[NUMBER_BALL * 2 + i + 1] * elastic;
			s[i + 1] = R;
		}
		if (s[i + 1] >= SCREEN_HEIGHT - R)
		{
			s[NUMBER_BALL * 2 + i + 1] = -s[NUMBER_BALL * 2 + i + 1] * elastic;
			s[i + 1] = SCREEN_HEIGHT - R;
		}
		for (int j = i + 2; j < NUMBER_BALL * 2; j += 2)
		{
			GLfloat dx = s[j] - s[i];
			GLfloat dy = s[j + 1] - s[i + 1];
			GLfloat d = sqrt(dx*dx + dy * dy);
			GLfloat dvx = s[NUMBER_BALL * 2 + j] - s[NUMBER_BALL * 2 + i];
			GLfloat dvy = s[NUMBER_BALL * 2 + j + 1] - s[NUMBER_BALL * 2 + i + 1];
			GLfloat Vxj = s[NUMBER_BALL * 2 + j];
			GLfloat Vyj = s[NUMBER_BALL * 2 + j + 1];
			GLfloat Vxi = s[NUMBER_BALL * 2 + i];
			GLfloat Vyi = s[NUMBER_BALL * 2 + i + 1];
			if (d <= 2 * R)
			{
				GLfloat midx = 0.5*(s[j] + s[i]);
				GLfloat midy = 0.5*(s[j + 1] + s[i + 1]);
				s[j] = (s[j] - midx) * 2 * R / d + midx;
				s[j + 1] = (s[j + 1] - midy) * 2 * R / d + midy;
				s[i] = (s[i] - midx) * 2 * R / d + midx;
				s[i + 1] = (s[i + 1] - midy) * 2 * R / d + midy;
				Vxj = s[NUMBER_BALL * 2 + j] - (dx)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
				Vyj = s[NUMBER_BALL * 2 + j + 1] - (dy)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
				Vxi = s[NUMBER_BALL * 2 + i] - (-dx)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
				Vyi = s[NUMBER_BALL * 2 + i + 1] - (-dy)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
				s[NUMBER_BALL * 2 + j] = Vxj * elastic;
				s[NUMBER_BALL * 2 + j + 1] = Vyj * elastic;
				s[NUMBER_BALL * 2 + i] = Vxi * elastic;
				s[NUMBER_BALL * 2 + i + 1] = Vyi * elastic;
			}
		}
		__syncthreads();

		X[i] = s[i];
		X[i + 1] = s[i + 1];
		V[i] = s[i + NUMBER_BALL * 2];
		V[i + 1] = s[i + 1 + NUMBER_BALL * 2];
	}
}
__global__
void collisionCuda(GLfloat *X, GLfloat*V)
{
	int i = (blockIdx.x * blockDim.x+threadIdx.x )* 2;
	for (int j = i + 2; j < NUMBER_BALL * 2; j += 2)
	{
		GLfloat dx = X[j] - X[i];
		GLfloat dy = X[j + 1] - X[i + 1];
		GLfloat d = sqrt(dx*dx + dy * dy);
		GLfloat dvx = V[j] - V[i];
		GLfloat dvy = V[j + 1] - V[i + 1];
		GLfloat Vxj = V[j];
		GLfloat Vyj = V[j + 1];
		GLfloat Vxi = V[i];
		GLfloat Vyi = V[i + 1];
		if (d <= 2 * R)
		{
			GLfloat midx = 0.5*(X[j] + X[i]);
			GLfloat midy = 0.5*(X[j + 1] + X[i + 1]);
			X[j] = (X[j] - midx) * 2 * R / d + midx;
			X[j + 1] = (X[j + 1] - midy) * 2 * R / d + midy;
			X[i] = (X[i] - midx) * 2 * R / d + midx;
			X[i + 1] = (X[i + 1] - midy) * 2 * R / d + midy;
			Vxj = V[j] - (dx)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
			Vyj = V[j + 1] - (dy)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
			Vxi = V[i] - (-dx)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
			Vyi = V[i + 1] - (-dy)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
			V[j] = Vxj * elastic;
			V[j + 1] = Vyj * elastic;
			V[i] = Vxi * elastic;
			V[i + 1] = Vyi * elastic;
		}
	}
	if (X[i] <= R)
	{
		V[i] = -V[i] * elastic;
		X[i] = R;
	}
	if (X[i] >= SCREEN_WIDTH - R)
	{
		V[i] = -V[i] * elastic;
		X[i] = SCREEN_WIDTH - R;
	}
	if (X[i + 1] <= R)
	{
		V[i + 1] = -V[i + 1] * elastic;
		X[i + 1] = R;
	}
	if (X[i + 1] >= SCREEN_HEIGHT - R)
	{
		V[i + 1] = -V[i + 1] * elastic;
		X[i + 1] = SCREEN_HEIGHT - R;
	}
}
/*void collision(GLfloat X[], GLfloat V[])
{
	for (int i = 0; i < NUMBER_BALL * 2; i += 2)
	{
		for (int j = i + 2; j < NUMBER_BALL * 2; j += 2)
		{
			GLfloat dx = X[j] - X[i];
			GLfloat dy = X[j + 1] - X[i + 1];
			GLfloat d = sqrt(dx*dx + dy * dy);
			GLfloat dvx = V[j] - V[i];
			GLfloat dvy = V[j + 1] - V[i + 1];
			if (d <= 2 * R)
			{
				GLfloat midx = 0.5*(X[j] + X[i]);
				GLfloat midy = 0.5*(X[j + 1] + X[i + 1]);
				X[j] = (X[j] - midx) * 2 * R / d + midx;
				X[j + 1] = (X[j + 1] - midy) * 2 * R / d + midy;
				X[i] = (X[i] - midx) * 2 * R / d + midx;
				X[i + 1] = (X[i + 1] - midy) * 2 * R / d + midy;
				GLfloat Vxj = V[j] - (dx)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
				GLfloat Vyj = V[j + 1] - (dy)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
				GLfloat Vxi = V[i] - (-dx)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
				GLfloat Vyi = V[i + 1] - (-dy)*(dvx*dx + dvy * dy) / (dx*dx + dy * dy);
				V[j] = Vxj * elastic;
				V[j + 1] = Vyj * elastic;
				V[i] = Vxi * elastic;
				V[i + 1] = Vyi * elastic;
			}
		}
		if (X[i] <= R)
		{
			V[i] = -V[i] * elastic;
			X[i] = R;
		}
		if (X[i] >= SCREEN_WIDTH - R)
		{
			V[i] = -V[i] * elastic;
			X[i] = SCREEN_WIDTH - R;
		}
		if (X[i + 1] <= R)
		{
			V[i + 1] = -V[i + 1] * elastic;
			X[i + 1] = R;
		}
		if (X[i + 1] >= SCREEN_HEIGHT - R)
		{
			V[i + 1] = -V[i + 1] * elastic;
			X[i + 1] = SCREEN_HEIGHT - R;
		}
	}
}*/
__global__
void gravityCudaShared(GLfloat *X, GLfloat *V)
{
	__shared__ GLfloat s[NUMBER_BALL * 4];
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	if (i < NUMBER_BALL * 2)
	{
		s[i] = X[i];
		s[i + 1] = X[i + 1];
		s[i + NUMBER_BALL * 2] = V[i];
		s[i + 1 + NUMBER_BALL * 2] = V[i + 1];

		GLfloat newXx = 0;
		GLfloat newXy = 0;
		GLfloat dx = 0;
		GLfloat dy = 0;
		GLfloat d = 0;
		GLfloat ax = 0;
		GLfloat ay = 0;
		for (int j = i + 2; j < NUMBER_BALL * 2; j += 2)
		{
			dx = s[j] - s[i];
			dy = s[j + 1] - s[i + 1];
			d = sqrt(dx*dx + dy * dy);
			ax += dx / (d*d*d);
			ay += dy / (d*d*d);
		}
		for (int j = 0; j < i; j += 2)
		{
			dx = s[j] - s[i];
			dy = s[j + 1] - s[i + 1];
			d = sqrt(dx*dx + dy * dy);
			ax += dx / (d*d*d);
			ay += dy / (d*d*d);
		}
		ax *= G * M;
		ay *= G * M;
		newXx = s[i] + s[NUMBER_BALL * 2 + i] + 0.5*ax*t*t;
		newXy = s[i + 1] + s[NUMBER_BALL * 2 + i + 1] + 0.5*ay*t*t;
		V[i] = s[NUMBER_BALL * 2 + i] + ax * t;
		V[i + 1] = s[NUMBER_BALL * 2 + i + 1] + ay * t;

		__syncthreads();
		X[i] = newXx;
		X[i + 1] = newXy;
	}
}
__global__
void gravityCuda(GLfloat *X, GLfloat *V)
{
	GLfloat newXx = 0;
	GLfloat newXy = 0;
	GLfloat dx = 0;
	GLfloat dy = 0;
	GLfloat d = 0;
	GLfloat ax = 0;
	GLfloat ay = 0;
	int index = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	for (int j=index+2;j<NUMBER_BALL*2;j+=2)
	{
			dx = X[j] - X[index];
			dy = X[j + 1] - X[index + 1];
			d = sqrt(dx*dx + dy * dy);
			ax += dx / (d*d*d);
			ay += dy / (d*d*d);
	}
	for (int j = 0; j < index; j += 2)
	{
		dx = X[j] - X[index];
		dy = X[j + 1] - X[index + 1];
		d = sqrt(dx*dx + dy * dy);
		ax += dx / (d*d*d);
		ay += dy / (d*d*d);
	}
	ax *= G * M;
	ay *= G * M;
	newXx=X[index]+V[index]+0.5*ax*t*t;
	newXy = X[index+1]+V[index+1]+0.5*ay*t*t;
	V[index] = V[index] + ax * t;
	V[index + 1] = V[index + 1] + ay * t;

	__syncthreads();
	X[index] = newXx;
	X[index+1] = newXy;

}
/*void gravity(GLfloat X[], GLfloat V[])
{
	GLfloat dx = 0;
	GLfloat dy = 0;
	GLfloat d = 0;
	GLfloat ax = 0;
	GLfloat ay = 0;
	GLfloat newX[NUMBER_BALL * 2] = { 0 };
	for (int i = 0; i < NUMBER_BALL * 2; i += 2)
	{
		ax = 0;
		ay = 0;
		for (int j = 0; j < NUMBER_BALL * 2; j += 2)
		{
			if (j != i)
			{
				dx = X[j] - X[i];
				dy = X[j + 1] - X[i + 1];
				d = sqrt(dx*dx + dy * dy);
				ax += dx / (d*d*d);
				ay += dy / (d*d*d);
			}
		}

		ax *= G * M;
		ay *= G * M;
		newX[i] = X[i] + V[i] + 0.5*ax*t*t;
		newX[i + 1] = X[i + 1] + V[i + 1] + 0.5*ay*t*t;
		V[i] = V[i] + ax * t;
		V[i + 1] = V[i + 1] + ay * t;
	}
	for (int i = 0; i < NUMBER_BALL * 2; i++)
	{
		X[i] = newX[i];
	}
}*/

__global__
void add(int n, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}
int main(void)
{
	GLFWwindow *window;
	// Initialize the library
	if (!glfwInit())
	{
		return -1;
	}

	// Create a windowed mode window and its OpenGL context
	window = glfwCreateWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "Hello World", NULL, NULL);

	if (!window)
	{
		glfwTerminate();
		return -1;
	}

	// Make the window's context current
	glfwMakeContextCurrent(window);
	glViewport(0.0f, 0.0f, SCREEN_WIDTH, SCREEN_HEIGHT); // specifies the part of the window to which OpenGL will draw (in pixels), convert from normalised to pixels
	glMatrixMode(GL_PROJECTION); // projection matrix defines the properties of the camera that views the objects in the world coordinate frame. Here you typically set the zoom factor, aspect ratio and the near and far clipping planes
	glLoadIdentity(); // replace the current matrix with the identity matrix and starts us a fresh because matrix transforms such as glOrpho and glRotate cumulate, basically puts us at (0, 0, 0)
	glOrtho(0, SCREEN_WIDTH, 0, SCREEN_HEIGHT, 0, 1); // essentially set coordinate system
	glMatrixMode(GL_MODELVIEW); // (default matrix mode) modelview matrix defines how your objects are transformed (meaning translation, rotation and scaling) in your world
	glLoadIdentity(); // same as above comment

	float *X, *V;
	// Allocate Unified Memory � accessible from CPU or GPU
	hipMallocManaged(&X, 2 * NUMBER_BALL * sizeof(float));
	hipMallocManaged(&V, 2 * NUMBER_BALL * sizeof(float));
	init(X);
	initV(V);
	int blockSize = (NUMBER_BALL + 31) / 32 * 32;
	int numBlocks = (NUMBER_BALL + blockSize - 1) / blockSize;

	// Loop until the user closes the window
	while (!glfwWindowShouldClose(window))
	{
		auto start = high_resolution_clock::now();
		///////global memory 
		//collisionCuda << <numBlocks, blockSize >> > (X, V); 
		//gravityCuda << <numBlocks, blockSize >> > (X, V); 
		///////shared memory
		collisionCudaShared << <numBlocks, blockSize >> > (X, V); 
		gravityCudaShared << <numBlocks, blockSize >> > (X, V); 
		hipDeviceSynchronize();
		auto stop = high_resolution_clock::now();
		auto duration = duration_cast<milliseconds>(stop - start);
		cout << "cost: " << duration.count() << " ms" << endl;

		glClear(GL_COLOR_BUFFER_BIT);
		// render OpenGL here
		glEnable(GL_POINT_SMOOTH);
		glEnableClientState(GL_VERTEX_ARRAY);
		
		glVertexPointer(2, GL_FLOAT, 0, X);
		glPointSize(ballsize);

		glDrawArrays(GL_POINTS, 0, NUMBER_BALL);
		glDisableClientState(GL_VERTEX_ARRAY);
		glDisable(GL_POINT_SMOOTH);
		// Swap front and back buffers
		glfwSwapBuffers(window);

		// Poll for and process events
		glfwPollEvents();
		//Sleep(20);
	}

	glfwTerminate();
	hipFree(X);
	hipFree(V);
	return 0;
}
